#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include "matrix.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__constant__ char global_matrixes_data[3 * sizeof(Matrix<3, 3>)];

void save_image_array(uint8_t* image_array, int width, int height, int channels) {
    /*
     * Save the data of an (RGB) image as a pixel map.
     * 
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     * 
     */            
    // Try opening the file
    FILE *imageFile;
    imageFile=fopen("./output_image.ppm","wb");
    if(imageFile==NULL){
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }
    
    // Configure the file
    fprintf(imageFile,"P6\n");               // P6 filetype
    fprintf(imageFile,"%d %d\n", width, height);      // dimensions
    fprintf(imageFile,"255\n");              // Max pixel
    
    // Write the image
    fwrite(image_array, 1, width * height * channels, imageFile);
    
    // Close the file
    fclose(imageFile);
}


void save_black_white_image(uint8_t* image_array, int width, int height) {
    uint8_t* color_image = (uint8_t*)malloc(width * height * 3);
    for (int i = 0; i < width * height; i++) {
        color_image[i * 3] = image_array[i];
        color_image[i * 3 + 1] = image_array[i];
        color_image[i * 3 + 2] = image_array[i];
    }
    save_image_array(color_image, width, height, 3);
    free(color_image);
}


void setBottomLeftMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0},
        {0, 0.5, 0},
        {0, 0, 1},
    };

    matrix->setData(newData);
}

void setBottomRightMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0.5},
        {0, 0.5, 0},
        {0, 0, 1},
    };

    matrix->setData(newData);
}

void setTopMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0.25},
        {0, 0.5, 0.5},
        {0, 0, 1},
    };

    matrix->setData(newData);
}


Matrix<3, 3> create_random_scale_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {(double)rand() / RAND_MAX, 0, 0},
        {0, (double)rand() / RAND_MAX, 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_translation_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1, 0, (double)rand() / RAND_MAX},
        {0, 1, (double)rand() / RAND_MAX},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}

Matrix<3, 3> create_random_rotation_matrix() {
    Matrix<3, 3> matrix;
    double angle = ((double)rand() / RAND_MAX) * 2 * M_PI;
    double data[3][3] = {
        {cos(angle), -sin(angle), 0},
        {sin(angle), cos(angle), 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_shear_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1, (double)rand() / RAND_MAX, 0},
        {(double)rand() / RAND_MAX, 1, 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_affine_matrix() {
    return create_random_scale_matrix() * create_random_rotation_matrix()* create_random_translation_matrix();
}


Matrix<3,3>* get_random_point(Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    int random = rand() % 3;
    if (random < 1) {
        return bottomLeftMatrix;
    } else if (random < 2) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}

__device__ Matrix<3,3>* get_random_point_gpu(hiprandState* state, Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    float random = hiprand_uniform(state);
    if (random < 1.0f / 3.0f) {
        return bottomLeftMatrix;
    } else if (random < 2.0f / 3.0f) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}



void create_triangle(Matrix<3, 1>* points, int amount, int iterations,Matrix<3, 1>* buffer ,Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    // the tirnalgle is on a one by one grid
    for (int j = 0; j < iterations; j++) {
        for (int i = 0; i < amount; i++) {
            Matrix<3, 3>* random_trig_point = get_random_point(bottomLeftMatrix, bottomRightMatrix, topMatrix);
            Matrix<3,1>* current_point = points + i;
            Matrix<3,1>* buffer_point = buffer + i;
            random_trig_point->mult(current_point, buffer_point);
            *current_point = *buffer_point;
        }
    }

}

Matrix<3, 1>* generate_random_points(int amount) {
    Matrix<3, 1>* points = (Matrix<3, 1>*)malloc(amount * sizeof(Matrix<3, 1>));
    for (int i = 0; i < amount; i++) {
        double data[3][1] = {
            {(double)rand() / RAND_MAX},
            {(double)rand() / RAND_MAX},
            {1.0},
        };
        points[i].setData(data);
    }
    return points;
}

// between 0 and one in all directions
void rescale_points(Matrix<3, 1>* points, int amount) {
    double min_x = 1;
    double min_y = 1;
    double max_x = 0;
    double max_y = 0;
    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        if (current_point.at(0, 0) < min_x) {
            min_x = current_point.at(0, 0);
        }
        if (current_point.at(1, 0) < min_y) {
            min_y = current_point.at(1, 0);
        }
        if (current_point.at(0, 0) > max_x) {
            max_x = current_point.at(0, 0);
        }
        if (current_point.at(1, 0) > max_y) {
            max_y = current_point.at(1, 0);
        }
    }
    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        current_point.at(0, 0) = (current_point.at(0, 0) - min_x) / (max_x - min_x);
        current_point.at(1, 0) = (current_point.at(1, 0) - min_y) / (max_y - min_y);
        points[i] = current_point;
    }
}


uint8_t* scale_to_image(Matrix<3, 1>* points, int amount, int width, int height) {
    uint8_t* image_array = (uint8_t*)calloc(width * height, sizeof(uint8_t));

    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        int x = fminf(current_point.at(0, 0) * width, width - 1);
        int y = fminf(height - current_point.at(1, 0)* height, height - 1);
        if (x < 0) {
            x = 0;
        }
        if (y < 0) {
            y = 0;
        }
        image_array[y * width + x] = 255;
    }
    return image_array;
}

__global__ void create_triangle_gpu_kernel(Matrix<3, 1>* points, int amount, int iterations, Matrix<3, 1>* buffer, int seed) {
    Matrix<3, 3>* global_matrixes = (Matrix<3, 3>*)global_matrixes_data;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= amount) {
        return;
    }

    hiprandState state;
    hiprand_init(seed + idx, 0, 0, &state);
    Matrix<3, 3>* random_trig_point;
    Matrix<3, 1>* original_point = points + idx;




    Matrix<3, 1>* current_point = original_point;
    Matrix<3, 1>* buffer_point = buffer + idx;

    Matrix<3, 1>* temp = current_point;

    for (int j = 0; j < iterations; j++) {
        random_trig_point = get_random_point_gpu(&state, &global_matrixes[0], &global_matrixes[1], &global_matrixes[2]);
        random_trig_point->mult(current_point, buffer_point);
        // we move the pointers around instead of copying the data
        current_point = buffer_point;
        buffer_point = temp;
        temp = current_point;
    }
    if (iterations % 2 != 0) {
        // we have a uneven amount of operations done and such, te last result is in the buffer and not in points.
        // so we copy it over
        *original_point = *buffer_point;
    }
}

void save_black_white_image_with_name(uint8_t* image_array, int width, int height, const char* filename) {
    uint8_t* color_image = (uint8_t*)malloc(width * height * 3);
    for (int i = 0; i < width * height; i++) {
        color_image[i * 3] = image_array[i];
        color_image[i * 3 + 1] = image_array[i];
        color_image[i * 3 + 2] = image_array[i];
    }

    FILE *imageFile = fopen(filename, "wb");
    if (imageFile == NULL) {
        perror("Cannot open output file");
        exit(EXIT_FAILURE);
    }
    fprintf(imageFile, "P6\n%d %d\n255\n", width, height);
    fwrite(color_image, 1, width * height * 3, imageFile);
    fclose(imageFile);
    free(color_image);
}


// RNG Setup Kernel (now fixed)
__global__ void setup_rng_kernel(hiprandState* states, int amount, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= amount) return;
    hiprand_init(seed, idx, 0, &states[idx]);
}

// GPU Kernel using persistent RNG states
__global__ void create_triangle_gpu_kernel(
    Matrix<3, 1>* input,
    int amount,
    Matrix<3, 1>* output,
    hiprandState* rng_states)
{
    Matrix<3, 3>* global_matrixes = (Matrix<3, 3>*)global_matrixes_data;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= amount) return;

    hiprandState* state = &rng_states[idx];

    Matrix<3,3>* random_trig_point;
    Matrix<3,1>* current_point = input + idx;
    Matrix<3,1>* buffer_point = output + idx;

    random_trig_point = get_random_point_gpu(state, &global_matrixes[0], &global_matrixes[1], &global_matrixes[2]);
    random_trig_point->mult(current_point, buffer_point);
}

// Main function with fixed RNG setup
void create_triangle_gpu_with_frames(Matrix<3, 1>* host_points, int amount, int iterations,
    const Matrix<3,3>& bottomLeftMatrix,
    const Matrix<3,3>& bottomRightMatrix,
    const Matrix<3,3>& topMatrix,
    int width, int height)
{
    // Allocate GPU memory
    Matrix<3, 1>* gpu_a;
    Matrix<3, 1>* gpu_b;
    hipMalloc(&gpu_a, amount * sizeof(Matrix<3, 1>));
    hipMalloc(&gpu_b, amount * sizeof(Matrix<3, 1>));

    // Copy initial points
    hipMemcpy(gpu_a, host_points, amount * sizeof(Matrix<3, 1>), hipMemcpyHostToDevice);

    // Allocate and setup RNG states
    hiprandState* d_states;
    hipMalloc(&d_states, amount * sizeof(hiprandState));

    int threadsPerBlock = 256;
    int blocksPerGrid = (amount + threadsPerBlock - 1) / threadsPerBlock;

    // ✅ Pass `amount` to RNG setup kernel
    setup_rng_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_states, amount, time(NULL));
    hipDeviceSynchronize();

    // Buffer for saving frames
    Matrix<3, 1>* save_buffer = (Matrix<3, 1>*)malloc(amount * sizeof(Matrix<3, 1>));
    bool using_a_as_input = true;

    for (int j = 0; j < iterations; j++) {
        Matrix<3, 1>* input = using_a_as_input ? gpu_a : gpu_b;
        Matrix<3, 1>* output = using_a_as_input ? gpu_b : gpu_a;

        create_triangle_gpu_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            input, amount, output,
            d_states
        );
        hipDeviceSynchronize();

        // Copy for frame
        hipMemcpy(save_buffer, output, amount * sizeof(Matrix<3, 1>), hipMemcpyDeviceToHost);

        // Save image
        rescale_points(save_buffer, amount);

        uint8_t* image_array = scale_to_image(save_buffer, amount, width, height);
        char filename[64];
        snprintf(filename, sizeof(filename), "./vid_imgs/frame_%03d.ppm", j);
        save_black_white_image_with_name(image_array, width, height, filename);
        free(image_array);

        using_a_as_input = !using_a_as_input;
    }

    // Copy final result
    Matrix<3, 1>* final_output = using_a_as_input ? gpu_b : gpu_a;
    hipMemcpy(host_points, final_output, amount * sizeof(Matrix<3, 1>), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(gpu_a);
    hipFree(gpu_b);
    hipFree(d_states);
    free(save_buffer);
}

void create_triangle_gpu(Matrix<3, 1>* points, int amount, int iterations , const Matrix<3,3>& bottomLeftMatrix, const Matrix<3,3>& bottomRightMatrix, const Matrix<3,3>& topMatrix) {
    Matrix<3, 1>* gpu_points;
    Matrix<3, 1>* gpu_buffer;

    hipMalloc(&gpu_points, amount * sizeof(Matrix<3, 1>));
    hipMalloc(&gpu_buffer, amount * sizeof(Matrix<3, 1>));

    hipMemcpy(gpu_points, points, amount * sizeof(Matrix<3, 1>), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = (amount + threadsPerBlock - 1) / threadsPerBlock;
    printf("blocksPerGrid: %d\n", blocksPerGrid);

    create_triangle_gpu_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_points, amount, iterations, gpu_buffer, time(NULL));
    hipDeviceSynchronize();
    hipMemcpy(points, gpu_points, amount * sizeof(Matrix<3, 1>), hipMemcpyDeviceToHost);
    hipFree(gpu_points);
    hipFree(gpu_buffer);

}


int main() {
    srand(81);
    int width = 500;
    int height = 500;
    int image_size = width * height;


    Matrix<3, 3> randomMatrixOne = create_random_affine_matrix();
    Matrix<3, 3> randomMatrixTwo = create_random_affine_matrix();
    Matrix<3, 3> randomMatrixThree = create_random_affine_matrix();
    randomMatrixOne.print();
    Matrix<3, 3> matrixesArray[3] = {randomMatrixOne, randomMatrixTwo, randomMatrixThree};
    hipMemcpyToSymbol(HIP_SYMBOL(global_matrixes_data), &matrixesArray, sizeof(Matrix<3, 3>)*3);
    
    // Generate random points
    int amount = 1000000;
    printf("Generating random points...\n");
    Matrix<3, 1>* points = generate_random_points(amount);
    Matrix<3, 1>* buffer = (Matrix<3, 1>*)malloc(amount * sizeof(Matrix<3, 1>));
    Matrix<3,3> bottomLeftMatrix;
    Matrix<3,3> bottomRightMatrix;
    Matrix<3,3> topMatrix;
    setBottomLeftMatrix(&bottomLeftMatrix);
    setBottomRightMatrix(&bottomRightMatrix);
    setTopMatrix(&topMatrix);



    printf("done\n");
    printf("Creating triangle...");
    //create_triangle(points, amount, 200, buffer, &bottomLeftMatrix, &bottomRightMatrix, &topMatrix);
    // create_triangle_gpu(points, amount, 20, bottomLeftMatrix, bottomRightMatrix, topMatrix);
    create_triangle_gpu_with_frames(points, amount, 200, randomMatrixOne, randomMatrixTwo, randomMatrixThree, width, height);
    //create_triangle_gpu(points, amount, 1000, randomMatrixOne, randomMatrixTwo, randomMatrixThree);
    printf("done\n");
    printf("Rescaling points...");
    rescale_points(points, amount);
    printf("done\n");
    printf("Creating to image...");
    uint8_t* image_array = scale_to_image(points, amount, width, height);
    printf("done\n");
    
    
    // Save the image
    save_black_white_image(image_array, width, height);

    system("ffmpeg -y -framerate 5 -i ./vid_imgs/frame_%03d.ppm -c:v libx264 -pix_fmt yuv420p output.mp4");
    
    // Free the memory
    free(image_array);
    free(points);
    free(buffer);
    
    return 0;
}