#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include "matrix.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "stable_random.h"

# define OUR_PI		3.14159265358979323846	/* pi */
StableRandom stable_random;


__constant__ char global_matrixes_data[3 * sizeof(Matrix<3, 3>)];

void save_image_array(uint8_t* image_array, int width, int height, int channels) {
    /*
     * Save the data of an (RGB) image as a pixel map.
     * 
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     * 
     */            
    // Try opening the file
    FILE *imageFile;
    imageFile=fopen("./output_image.ppm","wb");
    if(imageFile==NULL){
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }
    
    // Configure the file
    fprintf(imageFile,"P6\n");               // P6 filetype
    fprintf(imageFile,"%d %d\n", width, height);      // dimensions
    fprintf(imageFile,"255\n");              // Max pixel
    
    // Write the image
    fwrite(image_array, 1, width * height * channels, imageFile);
    
    // Close the file
    fclose(imageFile);
}


void save_black_white_image(uint8_t* image_array, int width, int height) {
    uint8_t* color_image = (uint8_t*)malloc(width * height * 3);
    for (int i = 0; i < width * height; i++) {
        color_image[i * 3] = image_array[i];
        color_image[i * 3 + 1] = image_array[i];
        color_image[i * 3 + 2] = image_array[i];
    }
    save_image_array(color_image, width, height, 3);
    free(color_image);
}


void setBottomLeftMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0},
        {0, 0.5, 0},
        {0, 0, 1},
    };

    matrix->setData(newData);
}

void setBottomRightMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0.5},
        {0, 0.5, 0},
        {0, 0, 1},
    };

    matrix->setData(newData);
}

void setTopMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0.25},
        {0, 0.5, 0.5},
        {0, 0, 1},
    };

    matrix->setData(newData);
}


Matrix<3, 3> create_random_scale_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {(double)stablerand_next(&stable_random), 0, 0},
        {0, (double)stablerand_next(&stable_random), 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_translation_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1, 0, stablerand_next(&stable_random)},
        {0, 1, stablerand_next(&stable_random)},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}

Matrix<3, 3> create_random_rotation_matrix() {
    Matrix<3, 3> matrix;
    double angle = ((double)stablerand_next(&stable_random)) * 2 * OUR_PI;
    double data[3][3] = {
        {cos(angle), -sin(angle), 0},
        {sin(angle), cos(angle), 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_shear_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1, (double)stablerand_next(&stable_random), 0},
        {(double)stablerand_next(&stable_random), 1, 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_affine_matrix() {
    return create_random_scale_matrix() * create_random_rotation_matrix()* create_random_translation_matrix();
}


Matrix<3,3>* get_random_point(Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    int random = (int)floor(stablerand_next(&stable_random) * 81) % 3;
    if (random < 1) {
        return bottomLeftMatrix;
    } else if (random < 2) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}

__device__ Matrix<3,3>* get_random_point_gpu(hiprandState* state, Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    float random = hiprand_uniform(state);
    if (random < 1.0f / 3.0f) {
        return bottomLeftMatrix;
    } else if (random < 2.0f / 3.0f) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}



void create_triangle(Matrix<3, 1>* points, int amount, int iterations,Matrix<3, 1>* buffer ,Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    // the tirnalgle is on a one by one grid
    for (int j = 0; j < iterations; j++) {
        for (int i = 0; i < amount; i++) {
            Matrix<3, 3>* random_trig_point = get_random_point(bottomLeftMatrix, bottomRightMatrix, topMatrix);
            Matrix<3,1>* current_point = points + i;
            Matrix<3,1>* buffer_point = buffer + i;
            random_trig_point->mult(current_point, buffer_point);
            *current_point = *buffer_point;
        }
    }

}

Matrix<3, 1>* generate_random_points(int amount) {
    Matrix<3, 1>* points = (Matrix<3, 1>*)malloc(amount * sizeof(Matrix<3, 1>));
    for (int i = 0; i < amount; i++) {
        double data[3][1] = {
            {(double)stablerand_next(&stable_random)},
            {(double)stablerand_next(&stable_random)},
            {1.0},
        };
        points[i].setData(data);
    }
    return points;
}

// between 0 and one in all directions
void rescale_points(Matrix<3, 1>* points, int amount) {
    double min_x = 1;
    double min_y = 1;
    double max_x = 0;
    double max_y = 0;
    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        if (current_point.at(0, 0) < min_x) {
            min_x = current_point.at(0, 0);
        }
        if (current_point.at(1, 0) < min_y) {
            min_y = current_point.at(1, 0);
        }
        if (current_point.at(0, 0) > max_x) {
            max_x = current_point.at(0, 0);
        }
        if (current_point.at(1, 0) > max_y) {
            max_y = current_point.at(1, 0);
        }
    }
    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        current_point.at(0, 0) = (current_point.at(0, 0) - min_x) / (max_x - min_x);
        current_point.at(1, 0) = (current_point.at(1, 0) - min_y) / (max_y - min_y);
        points[i] = current_point;
    }
}

uint8_t* scale_to_image(Matrix<3, 1>* points, int amount, int width, int height) {
    int pointsPerPixel = amount / (width * height);
    printf("pointsPerPixel: %d\n", pointsPerPixel);
    if (pointsPerPixel < 1) {
        pointsPerPixel = 1;
    }



    uint8_t* image_array = (uint8_t*)calloc(width * height, sizeof(uint8_t));

    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        int x = fminf(current_point.at(0, 0) * width, width - 1);
        int y = fminf(height - current_point.at(1, 0)* height, height - 1);
        if (x < 0) {
            x = 0;
        }
        if (y < 0) {
            y = 0;
        }
        image_array[y * width + x] = min(255, image_array[y * width + x] + 255 / pointsPerPixel);
    }
    return image_array;
}

__global__ void create_triangle_gpu_kernel(Matrix<3, 1>* points, int amount, int iterations, Matrix<3, 1>* buffer, int seed) {
    Matrix<3, 3>* global_matrixes = (Matrix<3, 3>*)global_matrixes_data;
    // striding
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        hiprandState state;
        hiprand_init(seed + idx, 0, 0, &state);
        Matrix<3, 3>* random_trig_point;
        Matrix<3, 1>* original_point = points + idx;

        Matrix<3, 1>* current_point = original_point;
        Matrix<3, 1>* buffer_point = buffer + idx;

        Matrix<3, 1>* temp = current_point;

        for (int j = 0; j < iterations; j++) {
            random_trig_point = get_random_point_gpu(&state, &global_matrixes[0], &global_matrixes[1], &global_matrixes[2]);
            random_trig_point->mult(current_point, buffer_point);
            // we move the pointers around instead of copying the data
            current_point = buffer_point;
            buffer_point = temp;
            temp = current_point;
        }
        if (iterations % 2 != 0) {
            // we have a uneven amount of operations done and such, te last result is in the buffer and not in points.
            // so we copy it over
            *original_point = *buffer_point;
        }
    }
}

void save_black_white_image_with_name(uint8_t* image_array, int width, int height, const char* filename) {
    uint8_t* color_image = (uint8_t*)malloc(width * height * 3);
    for (int i = 0; i < width * height; i++) {
        color_image[i * 3] = image_array[i];
        color_image[i * 3 + 1] = image_array[i];
        color_image[i * 3 + 2] = image_array[i];
    }

    FILE *imageFile = fopen(filename, "wb");
    if (imageFile == NULL) {
        perror("Cannot open output file");
        exit(EXIT_FAILURE);
    }
    fprintf(imageFile, "P6\n%d %d\n255\n", width, height);
    fwrite(color_image, 1, width * height * 3, imageFile);
    fclose(imageFile);
    free(color_image);
}


// RNG Setup Kernel (now fixed)
__global__ void setup_rng_kernel(hiprandState* states, int amount, int seed) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

// GPU Kernel using persistent RNG states
__global__ void create_triangle_gpu_kernel(
    Matrix<3, 1>* input,
    int amount,
    Matrix<3, 1>* output,
    hiprandState* rng_states)
{
    Matrix<3, 3>* global_matrixes = (Matrix<3, 3>*)global_matrixes_data;
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        hiprandState* state = &rng_states[idx];
        Matrix<3, 3>* random_trig_point;
        Matrix<3, 1>* current_point = input + idx;
        Matrix<3, 1>* buffer_point = output + idx;

        random_trig_point = get_random_point_gpu(state, &global_matrixes[0], &global_matrixes[1], &global_matrixes[2]);
        random_trig_point->mult(current_point, buffer_point);
    }
}

__global__ void create_image_floaty_gpu_kernel(Matrix<3, 1>* points, int amount, float* image_data, int width, int height,
    float* min_x, float* min_y, float* max_x, float* max_y
) {
    // striding
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        Matrix<3, 1>* current_point = points + idx;
        float x_f_norm = ((float) current_point->at(0, 0) - *min_x) / (*max_x - *min_x);
        float y_f_nrom = ((float) current_point->at(1, 0) - *min_y) / (*max_y - *min_y);
        float x_f = fminf(x_f_norm * width, width - 1);
        float y_f = fminf(height - y_f_nrom * height, height - 1);
        if (x_f < 0) {
            x_f = 0;
        }
        if (y_f < 0) {
            y_f = 0;
        }

        int x_i = (int)x_f;
        int y_i = (int)y_f;

        float dx = x_f - x_i;
        float dy = y_f - y_i;

        float w1 = (1.0f - dx) * (1.0f - dy);
        float w2 = dx * (1.0f - dy);
        float w3 = (1.0f - dx) * dy;
        float w4 = dx * dy;

        int x0 = x_i;
        int y0 = y_i;
        int x1 = min(x_i + 1, width - 1);
        int y1 = min(y_i + 1, height - 1);
        float mult = 5;
        atomicAdd(&image_data[y0 * width + x0], w1 * mult);
        atomicAdd(&image_data[y0 * width + x1], w2 * mult);
        atomicAdd(&image_data[y1 * width + x0], w3 * mult);
        atomicAdd(&image_data[y1 * width + x1], w4 * mult);

    }
}

__global__ void float_to_uint8_kernel(float* input, uint8_t* output, unsigned long size) {
    for (unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += blockDim.x * gridDim.x) {
        output[idx] = (uint8_t)fminf(input[idx], 255.0f);
        //output[idx] = 255;
    }
}

__device__ float fatomicMin(float *addr, float value)

{

        float old = *addr, assumed;

        if(old <= value) return old;

        do

        {

                assumed = old;

                old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));

        }while(old!=assumed);

        return old;

}

__device__ float fatomicMax(float *addr, float value)

{

        float old = *addr, assumed;

        if(old >= value) return old;

        do

        {

                assumed = old;

                old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));

        }while(old!=assumed);

        return old;

}


__global__ void get_scaling_params_kernel(Matrix<3, 1>* points, int amount, float* min_x, float* min_y, float* max_x, float* max_y) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        Matrix<3, 1>* current_point = points + idx;
        fatomicMin(min_x, (float) current_point->at(0, 0));
        fatomicMin(min_y, (float) current_point->at(1, 0));
        fatomicMax(max_x, (float) current_point->at(0, 0));
        fatomicMax(max_y, (float) current_point->at(1, 0));
    }
}

// Main function with fixed RNG setup
void create_triangle_gpu_with_frames(Matrix<3, 1>* host_points, int amount, int iterations,
    const Matrix<3,3>& bottomLeftMatrix,
    const Matrix<3,3>& bottomRightMatrix,
    const Matrix<3,3>& topMatrix,
    int width, int height)
{
    hipStream_t stream;
    hipStreamCreate(&stream);


    // Allocate GPU memory
    Matrix<3, 1>* gpu_a;
    Matrix<3, 1>* gpu_b;

    hipMallocAsync(&gpu_a, amount * sizeof(Matrix<3, 1>), stream);
    hipMallocAsync(&gpu_b, amount * sizeof(Matrix<3, 1>), stream);

    // Copy initial points
    hipMemcpyAsync(gpu_a, host_points, amount * sizeof(Matrix<3, 1>), hipMemcpyHostToDevice, stream);

    // Allocate and setup RNG states
    hiprandState* d_states;
    hipMallocAsync(&d_states, amount * sizeof(hiprandState), stream);

    int threadsPerBlock = 256;
    int blocksPerGrid = (amount + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = min(blocksPerGrid, 65535); // Limit to 65535 blocks

    // ✅ Pass `amount` to RNG setup kernel
    setup_rng_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_states, amount, time(NULL));
    //hipDeviceSynchronize();

    float* gpu_floaty_save_buffer = nullptr;
    hipMallocAsync(&gpu_floaty_save_buffer, width * height * sizeof(float), stream);
    hipMemsetAsync(gpu_floaty_save_buffer, 0, width * height * sizeof(float), stream);

    uint8_t* gpu_save_buffer = nullptr;
    hipMallocAsync(&gpu_save_buffer, width * height * sizeof(uint8_t), stream);
    hipMemsetAsync(gpu_save_buffer, 0, width * height * sizeof(uint8_t), stream);


    float* scaling_data = nullptr;
    hipMallocAsync(&scaling_data, 4 * sizeof(float), stream);
    float* min_x = scaling_data;
    float* min_y = scaling_data + 1;
    float* max_x = scaling_data + 2;
    float* max_y = scaling_data + 3;

    uint8_t* save_buffer = (uint8_t*)malloc(width * height * sizeof(uint8_t));

    bool using_a_as_input = true;

    // changed ordering so that the gpu is busy when the cpu is busy
    for (int j = 0; j < iterations; j++) {
        Matrix<3, 1>* input = using_a_as_input ? gpu_a : gpu_b;
        Matrix<3, 1>* output = using_a_as_input ? gpu_b : gpu_a;



        create_triangle_gpu_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            input, amount, output,
            d_states
        );
        //hipDeviceSynchronize();
        hipMemsetAsync(min_x, 1, sizeof(float), stream);
        hipMemsetAsync(min_y, 1, sizeof(float), stream);
        hipMemsetAsync(max_x, 0, sizeof(float), stream);
        hipMemsetAsync(max_y, 0, sizeof(float), stream);
        get_scaling_params_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            output, amount, min_x, min_y, max_x, max_y
        );
        create_image_floaty_gpu_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            output, amount, gpu_floaty_save_buffer, width, height,
            min_x, min_y, max_x, max_y
        );

        float_to_uint8_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            gpu_floaty_save_buffer, gpu_save_buffer, width * height
        );
        hipMemsetAsync(gpu_floaty_save_buffer, 0, width * height * sizeof(float), stream);


        // Copy for frame
        hipMemcpyAsync(save_buffer, gpu_save_buffer, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost, stream);

        // Save image
        //rescale_points(save_buffer, amount);

        //uint8_t* image_array = scale_to_image(save_buffer, amount, width, height);
        char filename[64];
        snprintf(filename, sizeof(filename), "./vid_imgs/frame_%03d.ppm", j);
        save_black_white_image_with_name(save_buffer, width, height, filename);
        //free(image_array);

        using_a_as_input = !using_a_as_input;
    }

    // Copy final result
    Matrix<3, 1>* final_output = using_a_as_input ? gpu_b : gpu_a;
    hipMemcpyAsync(host_points, final_output, amount * sizeof(Matrix<3, 1>), hipMemcpyDeviceToHost, stream);

    // Cleanup
    hipFreeAsync(gpu_a, stream);
    hipFreeAsync(gpu_b, stream);
    hipFreeAsync(d_states, stream);
    hipFreeAsync(gpu_floaty_save_buffer, stream);
    hipFreeAsync(gpu_save_buffer, stream);
    hipFreeAsync(scaling_data, stream);
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    free(save_buffer);
}

void create_triangle_gpu(Matrix<3, 1>* points, int amount, int iterations , const Matrix<3,3>& bottomLeftMatrix, const Matrix<3,3>& bottomRightMatrix, const Matrix<3,3>& topMatrix) {
    Matrix<3, 1>* gpu_points;
    Matrix<3, 1>* gpu_buffer;

    hipMalloc(&gpu_points, amount * sizeof(Matrix<3, 1>));
    hipMalloc(&gpu_buffer, amount * sizeof(Matrix<3, 1>));

    hipMemcpy(gpu_points, points, amount * sizeof(Matrix<3, 1>), hipMemcpyHostToDevice);
    unsigned long long kb = amount * sizeof(Matrix<3, 1>) / 1024;
    unsigned long long mb = kb / 1024;
    printf("Size of copy: %llu KB\n", kb);
    printf("Size of copy: %llu MB\n", mb);
    int threadsPerBlock = 256;
    int blocksPerGrid = (amount + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = min(blocksPerGrid, 65535); // Limit to 65535 blocks
    printf("blocksPerGrid: %d\n", blocksPerGrid);

    create_triangle_gpu_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_points, amount, iterations, gpu_buffer, time(NULL));
    hipDeviceSynchronize();
    hipMemcpy(points, gpu_points, amount * sizeof(Matrix<3, 1>), hipMemcpyDeviceToHost);
    hipFree(gpu_points);
    hipFree(gpu_buffer);

}


int main() {
    // 4321
    stablerand_init(&stable_random, 7878778);
    //srand(4321);
    int width = 1000;
    int height = 1000;
    int image_size = width * height;


    Matrix<3, 3> randomMatrixOne = create_random_affine_matrix();
    Matrix<3, 3> randomMatrixTwo = create_random_affine_matrix();
    Matrix<3, 3> randomMatrixThree = create_random_affine_matrix();
    randomMatrixOne.print();
    Matrix<3, 3> matrixesArray[3] = {randomMatrixOne, randomMatrixTwo, randomMatrixThree};
    hipMemcpyToSymbol(HIP_SYMBOL(global_matrixes_data), &matrixesArray, sizeof(Matrix<3, 3>)*3);
    
    // Generate random points
    int amount = 40000000;
    printf("Generating random points...\n");
    Matrix<3, 1>* points = generate_random_points(amount);
    Matrix<3, 1>* buffer = (Matrix<3, 1>*)malloc(amount * sizeof(Matrix<3, 1>));
    Matrix<3,3> bottomLeftMatrix;
    Matrix<3,3> bottomRightMatrix;
    Matrix<3,3> topMatrix;
    setBottomLeftMatrix(&bottomLeftMatrix);
    setBottomRightMatrix(&bottomRightMatrix);
    setTopMatrix(&topMatrix);



    printf("done\n");
    printf("Creating triangle...");
    //create_triangle(points, amount, 200, buffer, &bottomLeftMatrix, &bottomRightMatrix, &topMatrix);
    // create_triangle_gpu(points, amount, 20, bottomLeftMatrix, bottomRightMatrix, topMatrix);
    create_triangle_gpu_with_frames(points, amount, 200, randomMatrixOne, randomMatrixTwo, randomMatrixThree, width, height);
    //create_triangle_gpu(points, amount, 200, randomMatrixOne, randomMatrixTwo, randomMatrixThree);
    printf("done\n");
    printf("Rescaling points...");
    rescale_points(points, amount);
    printf("done\n");
    printf("Creating to image...");
    uint8_t* image_array = scale_to_image(points, amount, width, height);
    printf("done\n");
    
    
    // Save the image
    save_black_white_image(image_array, width, height);

    system("ffmpeg -y -framerate 5 -i ./vid_imgs/frame_%03d.ppm -c:v libx264 -pix_fmt yuv420p output.mp4");
    
    // Free the memory
    free(image_array);
    free(points);
    free(buffer);
    
    return 0;
}