#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include "matrix.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

void save_image_array(uint8_t* image_array, int width, int height, int channels) {
    /*
     * Save the data of an (RGB) image as a pixel map.
     * 
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     * 
     */            
    // Try opening the file
    FILE *imageFile;
    imageFile=fopen("./output_image.ppm","wb");
    if(imageFile==NULL){
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }
    
    // Configure the file
    fprintf(imageFile,"P6\n");               // P6 filetype
    fprintf(imageFile,"%d %d\n", width, height);      // dimensions
    fprintf(imageFile,"255\n");              // Max pixel
    
    // Write the image
    fwrite(image_array, 1, width * height * channels, imageFile);
    
    // Close the file
    fclose(imageFile);
}


void save_black_white_image(uint8_t* image_array, int width, int height) {
    uint8_t* color_image = (uint8_t*)malloc(width * height * 3);
    for (int i = 0; i < width * height; i++) {
        color_image[i * 3] = image_array[i];
        color_image[i * 3 + 1] = image_array[i];
        color_image[i * 3 + 2] = image_array[i];
    }
    save_image_array(color_image, width, height, 3);
    free(color_image);
}


void setBottomLeftMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0},
        {0, 0.5, 0},
        {0, 0, 1},
    };

    matrix->setData(newData);
}

void setBottomRightMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0.5},
        {0, 0.5, 0},
        {0, 0, 1},
    };

    matrix->setData(newData);
}

void setTopMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0.25},
        {0, 0.5, 0.5},
        {0, 0, 1},
    };

    matrix->setData(newData);
}


Matrix<3, 3> create_random_scale_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {(double)rand() / RAND_MAX, 0, 0},
        {0, (double)rand() / RAND_MAX, 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_translation_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1, 0, (double)rand() / RAND_MAX},
        {0, 1, (double)rand() / RAND_MAX},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}

Matrix<3, 3> create_random_rotation_matrix() {
    Matrix<3, 3> matrix;
    double angle = ((double)rand() / RAND_MAX) * 2 * M_PI;
    double data[3][3] = {
        {cos(angle), -sin(angle), 0},
        {sin(angle), cos(angle), 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_shear_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1, (double)rand() / RAND_MAX, 0},
        {(double)rand() / RAND_MAX, 1, 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_affine_matrix() {
    return create_random_scale_matrix() * create_random_translation_matrix();
}


Matrix<3,3>* get_random_point(Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    int random = rand() % 3;
    if (random < 1) {
        return bottomLeftMatrix;
    } else if (random < 2) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}

__device__ Matrix<3,3>* get_random_point_gpu(hiprandState* state, Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    float random = hiprand_uniform(state);
    if (random < 1.0f / 3.0f) {
        return bottomLeftMatrix;
    } else if (random < 2.0f / 3.0f) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}



void create_triangle(Matrix<3, 1>* points, int amount, int iterations,Matrix<3, 1>* buffer ,Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    // the tirnalgle is on a one by one grid
    for (int j = 0; j < iterations; j++) {
        for (int i = 0; i < amount; i++) {
            Matrix<3, 3>* random_trig_point = get_random_point(bottomLeftMatrix, bottomRightMatrix, topMatrix);
            Matrix<3,1>* current_point = points + i;
            Matrix<3,1>* buffer_point = buffer + i;
            random_trig_point->mult(current_point, buffer_point);
            *current_point = *buffer_point;
        }
    }

}

Matrix<3, 1>* generate_random_points(int amount) {
    Matrix<3, 1>* points = (Matrix<3, 1>*)malloc(amount * sizeof(Matrix<3, 1>));
    for (int i = 0; i < amount; i++) {
        double data[3][1] = {
            {(double)rand() / RAND_MAX},
            {(double)rand() / RAND_MAX},
            {1.0},
        };
        points[i].setData(data);
    }
    return points;
}

// between 0 and one in all directions
void rescale_points(Matrix<3, 1>* points, int amount) {
    double min_x = 1;
    double min_y = 1;
    double max_x = 0;
    double max_y = 0;
    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        if (current_point.at(0, 0) < min_x) {
            min_x = current_point.at(0, 0);
        }
        if (current_point.at(1, 0) < min_y) {
            min_y = current_point.at(1, 0);
        }
        if (current_point.at(0, 0) > max_x) {
            max_x = current_point.at(0, 0);
        }
        if (current_point.at(1, 0) > max_y) {
            max_y = current_point.at(1, 0);
        }
    }
    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        current_point.at(0, 0) = (current_point.at(0, 0) - min_x) / (max_x - min_x);
        current_point.at(1, 0) = (current_point.at(1, 0) - min_y) / (max_y - min_y);
        points[i] = current_point;
    }
}


uint8_t* scale_to_image(Matrix<3, 1>* points, int amount, int width, int height) {
    uint8_t* image_array = (uint8_t*)calloc(width * height, sizeof(uint8_t));

    for (int i = 0; i < amount; i++) {
        Matrix<3, 1> current_point = points[i];
        int x = fminf(current_point.at(0, 0) * width, width - 1);
        int y = fminf(height - current_point.at(1, 0)* height, height - 1);
        if (x < 0) {
            x = 0;
        }
        if (y < 0) {
            y = 0;
        }
        image_array[y * width + x] = 255;
    }
    return image_array;
}

__global__ void create_triangle_gpu_kernel(Matrix<3, 1>* points, int amount, int iterations, Matrix<3, 1>* buffer ,Matrix<3,3> bottomLeftMatrix, Matrix<3,3> bottomRightMatrix, Matrix<3,3> topMatrix, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= amount) {
        return;
    }

    hiprandState state;
    hiprand_init(seed + idx, 0, 0, &state);
    Matrix<3, 3>* random_trig_point;
    Matrix<3, 1>* original_point = points + idx;




    Matrix<3, 1>* current_point = original_point;
    Matrix<3, 1>* buffer_point = buffer + idx;

    Matrix<3, 1>* temp = current_point;

    for (int j = 0; j < iterations; j++) {
        random_trig_point = get_random_point_gpu(&state, &bottomLeftMatrix, &bottomRightMatrix, &topMatrix);
        random_trig_point->mult(current_point, buffer_point);
        // we move the pointers around instead of copying the data
        current_point = buffer_point;
        buffer_point = temp;
        temp = current_point;
    }
    if (iterations % 2 != 0) {
        // we have a uneven amount of operations done and such, te last result is in the buffer and not in points.
        // so we copy it over
        *original_point = *buffer_point;
    }
}


void create_triangle_gpu(Matrix<3, 1>* points, int amount, int iterations , const Matrix<3,3>& bottomLeftMatrix, const Matrix<3,3>& bottomRightMatrix, const Matrix<3,3>& topMatrix) {
    Matrix<3, 1>* gpu_points;
    Matrix<3, 1>* gpu_buffer;

    hipMalloc(&gpu_points, amount * sizeof(Matrix<3, 1>));
    hipMalloc(&gpu_buffer, amount * sizeof(Matrix<3, 1>));

    hipMemcpy(gpu_points, points, amount * sizeof(Matrix<3, 1>), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = (amount + threadsPerBlock - 1) / threadsPerBlock;
    printf("blocksPerGrid: %d\n", blocksPerGrid);

    create_triangle_gpu_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_points, amount, iterations, gpu_buffer, bottomLeftMatrix, bottomRightMatrix, topMatrix, time(NULL));
    hipDeviceSynchronize();
    hipMemcpy(points, gpu_points, amount * sizeof(Matrix<3, 1>), hipMemcpyDeviceToHost);
    hipFree(gpu_points);
    hipFree(gpu_buffer);

}


int main() {
    srand(42);
    int width = 1000;
    int height = 1000;
    int image_size = width * height;


    Matrix<3, 3> randomMatrixOne = create_random_affine_matrix();
    Matrix<3, 3> randomMatrixTwo = create_random_affine_matrix();
    Matrix<3, 3> randomMatrixThree = create_random_affine_matrix();
    randomMatrixOne.print();

    
    // Generate random points
    int amount = 10000000;
    printf("Generating random points...\n");
    Matrix<3, 1>* points = generate_random_points(amount);
    Matrix<3, 1>* buffer = (Matrix<3, 1>*)malloc(amount * sizeof(Matrix<3, 1>));
    Matrix<3,3> bottomLeftMatrix;
    Matrix<3,3> bottomRightMatrix;
    Matrix<3,3> topMatrix;
    setBottomLeftMatrix(&bottomLeftMatrix);
    setBottomRightMatrix(&bottomRightMatrix);
    setTopMatrix(&topMatrix);



    printf("done\n");
    printf("Creating triangle...");
    //create_triangle(points, amount, 200, buffer, &bottomLeftMatrix, &bottomRightMatrix, &topMatrix);
    create_triangle_gpu(points, amount, 1000, randomMatrixOne, randomMatrixTwo, randomMatrixThree);
    printf("done\n");
    printf("Rescaling points...");
    rescale_points(points, amount);
    printf("done\n");
    printf("Creating to image...");
    uint8_t* image_array = scale_to_image(points, amount, width, height);
    printf("done\n");
    
    
    // Save the image
    save_black_white_image(image_array, width, height);
    
    // Free the memory
    free(image_array);
    free(points);
    free(buffer);
    
    return 0;
}