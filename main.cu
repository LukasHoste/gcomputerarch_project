#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include "matrix.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "stable_random.h"
//#include <algorithm>
#include <array>
#include <random>
#include <chrono>

# define OUR_PI		3.14159265358979323846	/* pi */
# define OUR_E        2.71828182845904523536	/* e */
StableRandom stable_random;
#define WIDTH 1920
#define HEIGHT 1920
#define CHANNELS 3

struct ColoredPoint {
    Matrix<3, 1> pos;
    Matrix<4, 1> color;
};

template <typename T>
__host__ __device__ inline T my_clamp(const T& val, const T& lo, const T& hi) {
    return (val < lo) ? lo : (val > hi) ? hi : val;
}

__constant__ char global_matrixes_data[sizeof(Matrix<3, 3>) * 3 + sizeof(Matrix<4, 4>) * 3];

void save_image_array(uint8_t* image_array, int width, int height, int channels) {
    /*
     * Save the data of an (RGB) image as a pixel map.
     * 
     * Parameters:
     *  - param1: The data of an (RGB) image as a 1D array
     * 
     */            
    // Try opening the file
    FILE *imageFile;
    imageFile=fopen("./output_image.ppm","wb");
    if(imageFile==NULL){
        perror("ERROR: Cannot open output file");
        exit(EXIT_FAILURE);
    }
    
    // Configure the file
    fprintf(imageFile,"P6\n");               // P6 filetype
    fprintf(imageFile,"%d %d\n", width, height);      // dimensions
    fprintf(imageFile,"255\n");              // Max pixel
    
    // Write the image
    fwrite(image_array, 1, width * height * channels, imageFile);
    
    // Close the file
    fclose(imageFile);
}


void save_black_white_image(uint8_t* image_array, int width, int height) {
    uint8_t* color_image = (uint8_t*)malloc(width * height * 3);
    for (int i = 0; i < width * height; i++) {
        color_image[i * 3] = image_array[i];
        color_image[i * 3 + 1] = image_array[i];
        color_image[i * 3 + 2] = image_array[i];
    }
    save_image_array(color_image, width, height, 3);
    free(color_image);
}


void setBottomLeftMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0},
        {0, 0.5, 0},
        {0, 0, 1},
    };

    matrix->setData(newData);
}

void setBottomRightMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0.5},
        {0, 0.5, 0},
        {0, 0, 1},
    };

    matrix->setData(newData);
}

void setTopMatrix(Matrix<3,3>* matrix) {
    const double newData[3][3] = {
        {0.5, 0, 0.25},
        {0, 0.5, 0.5},
        {0, 0, 1},
    };

    matrix->setData(newData);
}


Matrix<3, 3> create_random_scale_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {(double)stablerand_next(&stable_random) *2 - 1, 0, 0},
        {0, (double)stablerand_next(&stable_random) * 2 - 1, 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_translation_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1, 0, stablerand_next(&stable_random) * 2 -1},
        {0, 1, stablerand_next(&stable_random) * 2 - 1},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}

Matrix<3, 3> create_random_rotation_matrix() {
    Matrix<3, 3> matrix;
    double angle = ((double)stablerand_next(&stable_random)) * 2 * OUR_PI - OUR_PI;
    double data[3][3] = {
        {cos(angle), -sin(angle), 0},
        {sin(angle), cos(angle), 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_shear_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1, (double)stablerand_next(&stable_random) * 2 - 1, 0},
        {(double)stablerand_next(&stable_random) * 2 -1, 1, 0},
        {0, 0, 1},
    };
    matrix.setData(data);
    return matrix;
}
Matrix<3, 3> create_random_affine_matrix() {
    return create_random_scale_matrix() * create_random_rotation_matrix()* create_random_translation_matrix();
}

// https://lisyarus.github.io/blog/posts/transforming-colors-with-matrices.html
Matrix<3, 3> create_random_darkening_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1 - (double)stablerand_next(&stable_random), 0, 0},
        {0, 1 - (double)stablerand_next(&stable_random), 0},
        {0, 0, 1 - (double)stablerand_next(&stable_random)}
    };
    matrix.setData(data);
    return matrix;
}

// Generates a small random double between -maxChange and +maxChange
double random_small_change(double maxChange = 0.1) {
    return ((double)stablerand_next(&stable_random)) * 2 * maxChange - maxChange;
}

// random decrease or increase of R, G, B with smaller range
Matrix<3, 3> create_random_decrease_increase_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1 + random_small_change(0.5), 0, 0},
        {0, 1 + random_small_change(0.5), 0},
        {0, 0, 1 + random_small_change(0.5)}
    };
    matrix.setData(data);
    return matrix;
}

// random color shift matrix e.g. R->G, G->B, B->R or R->B, G->R, B->G
Matrix<3, 3> create_color_shift_matrix_1() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {0, 0.8, 0},
        {0, 0, 0.8},
        {0.8, 0, 0}
    };
    matrix.setData(data);
    return matrix;
}

Matrix<3, 3> create_color_shift_matrix_2() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {0, 0, 0.8},
        {0, 0.8, 0},
        {0.8, 0, 0}
    };
    matrix.setData(data);
    return matrix;
}

void shuffle_array_in_place(int* arr, int size) {
    for (int i = size - 1; i > 0; i--) {
        int j = (int)(stablerand_next(&stable_random) * (i + 1));
        if (j != i) {
            int temp = arr[i];
            arr[i] = arr[j];
            arr[j] = temp;
        }
    }
}


// assume Matrix<3,3> has a setData(double[3][3]) member
Matrix<3, 3> create_random_color_shift(bool excludeIdentity = true) {
    // permutation array: index = destination channel 0=R,1=G,2=B
    // value = source channel to map from

    int perm[3] = {0, 1, 2};
    shuffle_array_in_place(perm, 3);
   
    
    // build the scaled permutation matrix
    double data[3][3] = {{0}};
    for (int dst = 0; dst < 3; ++dst) {
        int src = perm[dst];
        // Use your custom random function to scale between 0 and 2
        double scale = stablerand_next(&stable_random) * 2.0; // Replace with your function
        data[dst][src] = scale;
    }
    
    Matrix<3,3> M;
    M.setData(data);
    return M;
}

// lighten color
Matrix<3, 3> create_random_lighten_matrix() {
    Matrix<3, 3> matrix;
    double data[3][3] = {
        {1 + (double)stablerand_next(&stable_random), 0, 0},
        {0, 1 + (double)stablerand_next(&stable_random), 0},
        {0, 0, 1 + (double)stablerand_next(&stable_random)}
    };
    matrix.setData(data);
    return matrix;
}



Matrix<3,3>* get_random_point(Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    int random = (int)floor(stablerand_next(&stable_random) * 81) % 3;
    if (random < 1) {
        return bottomLeftMatrix;
    } else if (random < 2) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}

__device__ Matrix<3,3>* get_random_point_gpu(hiprandState* state, Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    float random = hiprand_uniform(state);
    if (random < 1.0f / 3.0f) {
        return bottomLeftMatrix;
    } else if (random < 2.0f / 3.0f) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}

__device__ Matrix<4,4>* get_random_point_gpu(hiprandState* state, Matrix<4,4>* bottomLeftMatrix, Matrix<4,4>* bottomRightMatrix, Matrix<4,4>* topMatrix) {
    float random = hiprand_uniform(state);
    if (random < 1.0f / 3.0f) {
        return bottomLeftMatrix;
    } else if (random < 2.0f / 3.0f) {
        return bottomRightMatrix;
    } else {
        return topMatrix;
    }
}



void create_triangle(Matrix<3, 1>* points, int amount, int iterations,Matrix<3, 1>* buffer ,Matrix<3,3>* bottomLeftMatrix, Matrix<3,3>* bottomRightMatrix, Matrix<3,3>* topMatrix) {
    // the tirnalgle is on a one by one grid
    for (int j = 0; j < iterations; j++) {
        for (int i = 0; i < amount; i++) {
            Matrix<3, 3>* random_trig_point = get_random_point(bottomLeftMatrix, bottomRightMatrix, topMatrix);
            Matrix<3,1>* current_point = points + i;
            Matrix<3,1>* buffer_point = buffer + i;
            random_trig_point->mult(current_point, buffer_point);
            *current_point = *buffer_point;
        }
    }

}


// between 0 and one in all directions
void rescale_points(ColoredPoint* points, int amount) {
    double min_x = 1;
    double min_y = 1;
    double max_x = 0;
    double max_y = 0;
    for (int i = 0; i < amount; i++) {
        ColoredPoint current_point = points[i];
        if (current_point.pos.at(0, 0) < min_x) {
            min_x = current_point.pos.at(0, 0);
        }
        if (current_point.pos.at(1, 0) < min_y) {
            min_y = current_point.pos.at(1, 0);
        }
        if (current_point.pos.at(0, 0) > max_x) {
            max_x = current_point.pos.at(0, 0);
        }
        if (current_point.pos.at(1, 0) > max_y) {
            max_y = current_point.pos.at(1, 0);
        }
    }
    for (int i = 0; i < amount; i++) {
        ColoredPoint current_point = points[i];
        current_point.pos.at(0, 0) = (current_point.pos.at(0, 0) - min_x) / (max_x - min_x);
        current_point.pos.at(1, 0) = (current_point.pos.at(1, 0) - min_y) / (max_y - min_y);
        points[i] = current_point;
    }
}

uint8_t* scale_to_image(ColoredPoint* points, int amount, int width, int height) {
    int pointsPerPixel = amount / (width * height);
    printf("pointsPerPixel: %d\n", pointsPerPixel);
    if (pointsPerPixel < 1) {
        pointsPerPixel = 1;
    }



    uint8_t* image_array = (uint8_t*)calloc(width * height * CHANNELS, sizeof(uint8_t));

    for (int i = 0; i < amount; i++) {
        ColoredPoint current_point = points[i];
        int x = fminf(current_point.pos.at(0, 0) * width, width - 1);
        int y = fminf(height - current_point.pos.at(1, 0)* height, height - 1);
        if (x < 0) {
            x = 0;
        }
        if (y < 0) {
            y = 0;
        }
        uint rIndex = (y * width + x) * 3;
        uint gIndex = rIndex + 1;
        uint bIndex = gIndex + 1;

        uint rValue = (uint)(current_point.color.at(0, 0) * 255);
        uint gValue = (uint)(current_point.color.at(1, 0) * 255);
        uint bValue = (uint)(current_point.color.at(2, 0) * 255);
    
        image_array[rIndex] = min(255, image_array[rIndex] + rValue / pointsPerPixel);
        image_array[gIndex] = min(255, image_array[gIndex] + gValue / pointsPerPixel);
        image_array[bIndex] = min(255, image_array[bIndex] + bValue / pointsPerPixel);
    }
    return image_array;
}

__global__ void create_triangle_gpu_kernel(ColoredPoint* points, int amount, int iterations, ColoredPoint* buffer, hiprandState* rng_states) {
    Matrix<3, 3>* global_matrixes = (Matrix<3, 3>*)global_matrixes_data;
    char* raw_color_matrixes = global_matrixes_data + 3 * sizeof(Matrix<3, 3>);

    Matrix<4, 4>* global_color_matr = (Matrix<4, 4>*) raw_color_matrixes;

    // striding
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        ColoredPoint* og_buffer = buffer + idx;


        hiprandState* state = &rng_states[idx];
        Matrix<3, 3>* random_pos_matrix;
        Matrix<4, 4>* random_color_matrix;
        ColoredPoint* original_point = points + idx;

        ColoredPoint* current_point = original_point;
        ColoredPoint* buffer_point = buffer + idx;

        ColoredPoint* temp = current_point;

        for (int j = 0; j < iterations; j++) {
            random_pos_matrix = get_random_point_gpu(state, &global_matrixes[0], &global_matrixes[1], &global_matrixes[2]);
            random_color_matrix = get_random_point_gpu(state, &global_color_matr[0], &global_color_matr[1], &global_color_matr[2]);
            //Matrix<3, 1> current_pos = current_point->pos;
            //buffer_point->pos = current_pos;

            random_pos_matrix->mult(&current_point->pos, &buffer_point->pos);
            random_color_matrix->mult(&current_point->color, &buffer_point->color);

            // color correcting
            /*buffer_point->color.at(2, 0) = 1.0;
            for(int i = 0; i < 2; i++) {
                buffer_point->color.at(i, 0) = fminf(fmaxf(buffer_point->color.at(i, 0), 0.0f), 1.0f);
            }*/

            // we move the pointers around instead of copying the data
            temp = current_point;
            current_point = buffer_point;
            buffer_point = temp;
        }
        /*og_buffer = *current_point;
        if (iterations % 2 != 0) {
            // we have a even amount of operations done and such, te last result is in the points and not in the buffer.
            // so we copy it over
            *og_buffer = *current_point;
        }*/
    }
}

void save_image_array(const char* filename, uint8_t* image_array) {
    FILE *imageFile = fopen(filename, "wb");
    if (!imageFile) {
        perror("Cannot open output file");
        exit(EXIT_FAILURE);
    }
    fprintf(imageFile, "P6\n%d %d\n255\n", WIDTH, HEIGHT);
    fwrite(image_array, 1, WIDTH * HEIGHT * CHANNELS, imageFile);
    fclose(imageFile);
}

ColoredPoint* generate_random_points(int amount) {
    ColoredPoint* points = new ColoredPoint[amount];
    for (int i = 0; i < amount; i++) {
        double pos_data[3][1] = {
            {(double)stablerand_next(&stable_random)},
            {(double)stablerand_next(&stable_random)},
            {1.0}
        };
        points[i].pos.setData(pos_data);
        
        // Start with white color
        double color_data[4][1] = {{(double)stablerand_next(&stable_random)}, {(double)stablerand_next(&stable_random)}, {(double)stablerand_next(&stable_random)}, {(double)stablerand_next(&stable_random)}};
        points[i].color.setData(color_data);
    }
    return points;
}

void save_image_with_name(uint8_t* image_array, int width, int height, const char* filename) {
    uint8_t* color_image = (uint8_t*)malloc(width * height * 3);
    for (int i = 0; i < width * height * CHANNELS; i++) {
        color_image[i] = image_array[i];
    }

    FILE *imageFile = fopen(filename, "wb");
    if (imageFile == NULL) {
        perror("Cannot open output file");
        exit(EXIT_FAILURE);
    }
    fprintf(imageFile, "P6\n%d %d\n255\n", width, height);
    fwrite(color_image, 1, width * height * 3, imageFile);
    fclose(imageFile);
    free(color_image);
}

__device__ float fatomicMin(float *addr, float value)

{

        float old = *addr, assumed;

        if(old <= value) return old;

        do

        {

                assumed = old;

                old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));

        }while(old!=assumed);

        return old;

}

__device__ float fatomicMax(float *addr, float value)

{

        float old = *addr, assumed;

        if(old >= value) return old;

        do

        {

                assumed = old;

                old = atomicCAS((unsigned int*)addr, __float_as_int(assumed), __float_as_int(value));

        }while(old!=assumed);

        return old;

}


// RNG Setup Kernel (now fixed)
__global__ void setup_rng_kernel(hiprandState* states, int amount, int seed) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

__global__ void create_image_floaty_gpu_kernel(ColoredPoint* points, int amount, float* image_data, int width, int height,
    float* min_x, float* min_y, float* max_x, float* max_y,
    float* min_r, float* min_g, float* min_b, float* max_r, float* max_g, float* max_b
) {
    // striding
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        ColoredPoint* current_point = points + idx;
        float x_f_norm = ((float) current_point->pos.at(0, 0) - *min_x) / (*max_x - *min_x);
        float y_f_nrom = ((float) current_point->pos.at(1, 0) - *min_y) / (*max_y - *min_y);

        float r_f_norm = ((float) current_point->color.at(0, 0) - *min_r) / (*max_r - *min_r);
        float g_f_norm = ((float) current_point->color.at(1, 0) - *min_g) / (*max_g - *min_g);
        float b_f_norm = ((float) current_point->color.at(2, 0) - *min_b) / (*max_b - *min_b);


        float x_f = fminf(x_f_norm * width, width - 1);
        float y_f = fminf(height - y_f_nrom * height, height - 1);
        if (x_f < 0) {
            x_f = 0;
        }
        if (y_f < 0) {
            y_f = 0;
        }

        int x_i = (int)x_f;
        int y_i = (int)y_f;

        float dx = x_f - x_i;
        float dy = y_f - y_i;

        float w1 = (1.0f - dx) * (1.0f - dy);
        float w2 = dx * (1.0f - dy);
        float w3 = (1.0f - dx) * dy;
        float w4 = dx * dy;

        int x0 = x_i;
        int y0 = y_i;
        int x1 = min(x_i + 1, width - 1);
        int y1 = min(y_i + 1, height - 1);
        float mult = 1;

        uint firstIndexR = (y0 * width + x0) * 3;
        uint firstIndexG = firstIndexR + 1;
        uint firstIndexB = firstIndexG + 1;

        uint secondIndexR = (y0 * width + x1) * 3;
        uint secondIndexG = secondIndexR + 1;
        uint secondIndexB = secondIndexG + 1;

        uint thirdIndexR = (y1 * width + x0) * 3;
        uint thirdIndexG = thirdIndexR + 1;
        uint thirdIndexB = thirdIndexG + 1;
        
        uint fourthIndexR = (y1 * width + x1) * 3;
        uint fourthIndexG = fourthIndexR + 1;
        uint fourthIndexB = fourthIndexG + 1;

        float rValue = r_f_norm;
        float gValue = g_f_norm;
        float bValue = b_f_norm;

        //rValue = 1;
        //gValue = 1;
        //bValue = 1;
        //w1 = 1;
        //w2 = 1;
        //w3 = 1;
        //w4 = 1;


        // atomic adds

        atomicAdd(&image_data[firstIndexR], w1 * rValue * mult);
        atomicAdd(&image_data[firstIndexG], w1 * gValue * mult);
        atomicAdd(&image_data[firstIndexB], w1 * bValue * mult);

        atomicAdd(&image_data[secondIndexR], w2 * rValue * mult);
        atomicAdd(&image_data[secondIndexG], w2 * gValue * mult);
        atomicAdd(&image_data[secondIndexB], w2 * bValue * mult);

        atomicAdd(&image_data[thirdIndexR], w3 * rValue * mult);
        atomicAdd(&image_data[thirdIndexG], w3 * gValue * mult);
        atomicAdd(&image_data[thirdIndexB], w3 * bValue * mult);

        atomicAdd(&image_data[fourthIndexR], w4 * rValue * mult);
        atomicAdd(&image_data[fourthIndexG], w4 * gValue * mult);
        atomicAdd(&image_data[fourthIndexB], w4 * bValue * mult);
    }
}
__device__ float mapFunction(float value, float pin) {
    //return value * 2 - value * value;
    //float o = powf(value, 0.1) * 0.4;
    float o = (1 - exp((-value *0.3 ))) * 0.9;
    //float o = (1 - exp(((value * 2) / pin) * logf(1- OUR_E / 3)))*1.01;
    //return value;
    if (o * 255 > 255) {
        return 1;
    }
    return o;
}


__global__ void float_to_uint8_kernel(float* input, uint8_t* output, unsigned long amount, float* min_r, float* min_g, float* min_b, float* max_r, float* max_g, float* max_b) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        float r = input[idx * 3];
        float g = input[idx * 3 + 1];
        float b = input[idx * 3 + 2];

        // clamped
        //r = (r - *min_r) / (*max_r - *min_r);
        //g = (g - *min_g) / (*max_g - *min_g);
        //b = (b - *min_b) / (*max_b - *min_b);



        output[idx * 3] = (uint8_t)(mapFunction(r, *max_r) * 255);
        output[idx * 3 + 1] = (uint8_t)(mapFunction(g, *max_g) * 255);
        output[idx * 3 + 2] = (uint8_t)(mapFunction(b, *max_b) * 255);
    }
}




__global__ void get_scaling_params_kernel(ColoredPoint* points, int amount, float* min_x, float* min_y, float* max_x, float* max_y,
    float* min_r, float* min_g, float* min_b, float* max_r, float* max_g, float* max_b) {
    
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        ColoredPoint* current_point = points + idx;
        fatomicMin(min_x, (float) current_point->pos.at(0, 0));
        fatomicMin(min_y, (float) current_point->pos.at(1, 0));
        fatomicMax(max_x, (float) current_point->pos.at(0, 0));
        fatomicMax(max_y, (float) current_point->pos.at(1, 0));
        
        fatomicMin(min_r, (float) current_point->color.at(0, 0));
        fatomicMin(min_g, (float) current_point->color.at(1, 0));
        fatomicMin(min_b, (float) current_point->color.at(2, 0));
        fatomicMax(max_r, (float) current_point->color.at(0, 0));
        fatomicMax(max_g, (float) current_point->color.at(1, 0));
        fatomicMax(max_b, (float) current_point->color.at(2, 0));
    }
}

__global__ void get_color_scaling_params(float* rgbValues, int amount, float* min_r, float* min_g, float* min_b, float* max_r, float* max_g, float* max_b) {
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < amount; idx += blockDim.x * gridDim.x) {
        fatomicMin(min_r, rgbValues[idx * 3]);
        fatomicMin(min_g, rgbValues[idx * 3 + 1]);
        fatomicMin(min_b, rgbValues[idx * 3 + 2]);
        fatomicMax(max_r, rgbValues[idx * 3]);
        fatomicMax(max_g, rgbValues[idx * 3 + 1]);
        fatomicMax(max_b, rgbValues[idx * 3 + 2]);
    }
}

// Main function with fixed RNG setup
void create_triangle_gpu_with_frames(ColoredPoint* host_points, int amount, int iterations,
    int width, int height)
{
    hipStream_t stream;
    hipStreamCreate(&stream);


    // Allocate GPU memory
    ColoredPoint* gpu_a;
    ColoredPoint* gpu_b;

    hipMallocAsync(&gpu_a, amount * sizeof(ColoredPoint), stream);
    hipMallocAsync(&gpu_b, amount * sizeof(ColoredPoint), stream);

    // Copy initial points
    hipMemcpyAsync(gpu_a, host_points, amount * sizeof(ColoredPoint), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(gpu_b, host_points, amount * sizeof(ColoredPoint), hipMemcpyHostToDevice, stream);

    // Allocate and setup RNG states
    hiprandState* d_states;
    hipMallocAsync(&d_states, amount * sizeof(hiprandState), stream);

    int threadsPerBlock = 256;
    int blocksPerGrid = (amount + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = min(blocksPerGrid, 65535); // Limit to 65535 blocks

    // ✅ Pass `amount` to RNG setup kernel
    setup_rng_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_states, amount, time(NULL));
    //hipDeviceSynchronize();

    float* gpu_floaty_save_buffer = nullptr;
    hipMallocAsync(&gpu_floaty_save_buffer, width * height * sizeof(float) * 3, stream);
    hipMemsetAsync(gpu_floaty_save_buffer, 0, width * height * sizeof(float) * 3, stream);

    uint8_t* gpu_save_buffer = nullptr;
    hipMallocAsync(&gpu_save_buffer, width * height * sizeof(uint8_t) * 3, stream);
    hipMemsetAsync(gpu_save_buffer, 0, width * height * sizeof(uint8_t) * 3, stream);


    float* scaling_data = nullptr;
    hipMallocAsync(&scaling_data, 10 * sizeof(float), stream);
    float* min_x = scaling_data;
    float* min_y = scaling_data + 1;
    float* max_x = scaling_data + 2;
    float* max_y = scaling_data + 3;
    
    float* min_r = scaling_data + 4;
    float* min_g = scaling_data + 5;
    float* min_b = scaling_data + 6;

    float* max_r = scaling_data + 7;
    float* max_g = scaling_data + 8;
    float* max_b = scaling_data + 9;

    uint8_t* save_buffer = (uint8_t*)malloc(width * height * sizeof(uint8_t) * 3);

    float fixed_min_start = 1000;
    float fixed_max_start = -1000;

    bool using_a_as_input = true;
    printf("starting \n");
    // changed ordering so that the gpu is busy when the cpu is busy
    for (int j = 0; j < iterations; j++) {
        ColoredPoint* input = using_a_as_input ? gpu_a : gpu_b;
        ColoredPoint* output = using_a_as_input ? gpu_b : gpu_a;



        create_triangle_gpu_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            input, amount, 1, output,
            d_states
        );

        ColoredPoint temp[10];
        hipMemcpyAsync(temp, output, 10 * sizeof(ColoredPoint), hipMemcpyDeviceToHost, stream);
        for (int i = 0; i < 10; i++) {
            //printf("temp[%d]: %f %f %f\n", i, temp[i].color.at(0, 0), temp[i].color.at(1, 0), temp[i].color.at(2, 0));
        }

        //hipDeviceSynchronize();

        hipMemcpyAsync(min_x, &fixed_min_start, sizeof(float),hipMemcpyHostToDevice,stream);
        hipMemcpyAsync(min_y, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(max_x, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(max_y, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice,stream);
        hipMemcpyAsync(min_r, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(min_g, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(min_b, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(max_r, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(max_g, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(max_b, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
        get_scaling_params_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            output, amount, min_x, min_y, max_x, max_y, min_r, min_g, min_b, max_r, max_g, max_b
        );
        float cpu_minr = 1;
        hipMemcpyAsync(&cpu_minr, min_r, sizeof(float), hipMemcpyDeviceToHost, stream);
        float cpu_maxr = 0;
        hipMemcpyAsync(&cpu_maxr, max_r, sizeof(float), hipMemcpyDeviceToHost, stream);
        float cpu_ming = 1;
        hipMemcpyAsync(&cpu_ming, min_g, sizeof(float), hipMemcpyDeviceToHost, stream);
        float cpu_maxg = 0;
        hipMemcpyAsync(&cpu_maxg, max_g, sizeof(float), hipMemcpyDeviceToHost, stream);
        float cpu_minb = 1;
        hipMemcpyAsync(&cpu_minb, min_b, sizeof(float), hipMemcpyDeviceToHost, stream);
        float cpu_maxb = 0;
        hipMemcpyAsync(&cpu_maxb, max_b, sizeof(float), hipMemcpyDeviceToHost, stream);

        printf("cpu_minr: %f\n", cpu_minr);
        printf("cpu_maxr: %f\n", cpu_maxr);
        printf("cpu_ming: %f\n", cpu_ming);
        printf("cpu_maxg: %f\n", cpu_maxg);
        printf("cpu_minb: %f\n", cpu_minb);
        printf("cpu_maxb: %f\n", cpu_maxb);

       

        create_image_floaty_gpu_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            output, amount, gpu_floaty_save_buffer, width, height,
            min_x, min_y, max_x, max_y,
            min_r, min_g, min_b, max_r, max_g, max_b
        );

        hipMemcpyAsync(min_r, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(min_g, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(min_b, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(max_r, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(max_g, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(max_b, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
        // get the scaling now that all is accumulated!
        get_color_scaling_params<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            gpu_floaty_save_buffer, width * height, min_r, min_g, min_b, max_r, max_g, max_b
        );
        

        float_to_uint8_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            gpu_floaty_save_buffer, gpu_save_buffer, width * height, min_r, min_g, min_b, max_r, max_g, max_b
        );
        hipMemsetAsync(gpu_floaty_save_buffer, 0, width * height * sizeof(float) * 3, stream);
        printf("starting memcopy\n");
        // Copy for frame
        hipMemcpyAsync(save_buffer, gpu_save_buffer, width * height * sizeof(uint8_t) * 3, hipMemcpyDeviceToHost, stream);

        // Save image
        //rescale_points(save_buffer, amount);

        //uint8_t* image_array = scale_to_image(save_buffer, amount, width, height);
        char filename[64];
        snprintf(filename, sizeof(filename), "./vid_imgs/frame_%03d.ppm", j);
        printf("saving image %s\n", filename);
        save_image_with_name(save_buffer, width, height, filename);
        printf("done\n");
        //free(image_array);

        using_a_as_input = !using_a_as_input;
    }

    // Copy final result
    ColoredPoint* final_output = using_a_as_input ? gpu_b : gpu_a;
    hipMemcpyAsync(host_points, final_output, amount * sizeof(Matrix<3, 1>), hipMemcpyDeviceToHost, stream);

    // Cleanup
    hipFreeAsync(gpu_a, stream);
    hipFreeAsync(gpu_b, stream);
    hipFreeAsync(d_states, stream);
    hipFreeAsync(gpu_floaty_save_buffer, stream);
    hipFreeAsync(gpu_save_buffer, stream);
    hipFreeAsync(scaling_data, stream);
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    free(save_buffer);
}


// Main function with fixed RNG setup
void create_triangle_gpu_single(ColoredPoint* host_points, int amount, int iterations,
    int width, int height, char* output_buffer, float* cpu_measured_time)
{
    auto start = std::chrono::high_resolution_clock::now();
    hipStream_t stream;
    hipStreamCreate(&stream);


    // Allocate GPU memory
    ColoredPoint* gpu_a;
    ColoredPoint* gpu_b;

    hipMallocAsync(&gpu_a, amount * sizeof(ColoredPoint), stream);
    hipMallocAsync(&gpu_b, amount * sizeof(ColoredPoint), stream);

    // Copy initial points
    hipMemcpyAsync(gpu_a, host_points, amount * sizeof(ColoredPoint), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(gpu_b, host_points, amount * sizeof(ColoredPoint), hipMemcpyHostToDevice, stream);

    // Allocate and setup RNG states
    hiprandState* d_states;
    hipMallocAsync(&d_states, amount * sizeof(hiprandState), stream);

    int threadsPerBlock = 256;
    int blocksPerGrid = (amount + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = min(blocksPerGrid, 65535); // Limit to 65535 blocks

    // ✅ Pass `amount` to RNG setup kernel
    setup_rng_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(d_states, amount, time(NULL));
    //hipDeviceSynchronize();

    float* gpu_floaty_save_buffer = nullptr;
    hipMallocAsync(&gpu_floaty_save_buffer, width * height * sizeof(float) * 3, stream);
    hipMemsetAsync(gpu_floaty_save_buffer, 0, width * height * sizeof(float) * 3, stream);

    uint8_t* gpu_save_buffer = nullptr;
    hipMallocAsync(&gpu_save_buffer, width * height * sizeof(uint8_t) * 3, stream);
    hipMemsetAsync(gpu_save_buffer, 0, width * height * sizeof(uint8_t) * 3, stream);


    float* scaling_data = nullptr;
    hipMallocAsync(&scaling_data, 10 * sizeof(float), stream);
    float* min_x = scaling_data;
    float* min_y = scaling_data + 1;
    float* max_x = scaling_data + 2;
    float* max_y = scaling_data + 3;
    
    float* min_r = scaling_data + 4;
    float* min_g = scaling_data + 5;
    float* min_b = scaling_data + 6;

    float* max_r = scaling_data + 7;
    float* max_g = scaling_data + 8;
    float* max_b = scaling_data + 9;

    float fixed_min_start = 1000;
    float fixed_max_start = -1000;

    // changed ordering so that the gpu is busy when the cpu is busy

    ColoredPoint* input = gpu_a;
    ColoredPoint* output = gpu_b;


    create_triangle_gpu_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        input, amount, iterations, output,
        d_states
    );

    ColoredPoint temp[10];
    hipMemcpyAsync(temp, output, 10 * sizeof(ColoredPoint), hipMemcpyDeviceToHost, stream);

    //hipDeviceSynchronize();

    hipMemcpyAsync(min_x, &fixed_min_start, sizeof(float),hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(min_y, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(max_x, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(max_y, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice,stream);
    hipMemcpyAsync(min_r, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(min_g, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(min_b, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(max_r, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(max_g, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(max_b, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
    get_scaling_params_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        output, amount, min_x, min_y, max_x, max_y, min_r, min_g, min_b, max_r, max_g, max_b
    );

    create_image_floaty_gpu_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        output, amount, gpu_floaty_save_buffer, width, height,
        min_x, min_y, max_x, max_y,
        min_r, min_g, min_b, max_r, max_g, max_b
    );

    hipMemcpyAsync(min_r, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(min_g, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(min_b, &fixed_min_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(max_r, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(max_g, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(max_b, &fixed_max_start, sizeof(float), hipMemcpyHostToDevice, stream);
    // get the scaling now that all is accumulated!
    get_color_scaling_params<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        gpu_floaty_save_buffer, width * height, min_r, min_g, min_b, max_r, max_g, max_b
    );
    

    float_to_uint8_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
        gpu_floaty_save_buffer, gpu_save_buffer, width * height, min_r, min_g, min_b, max_r, max_g, max_b
    );
    hipMemsetAsync(gpu_floaty_save_buffer, 0, width * height * sizeof(float) * 3, stream);
    // Copy for frame
    hipMemcpyAsync(output_buffer, gpu_save_buffer, width * height * sizeof(uint8_t) * 3, hipMemcpyDeviceToHost, stream);

    // Save image
    //rescale_points(save_buffer, amount);
    save_image_array("./vid_imgs/frame.ppm", (uint8_t*)output_buffer);
    //int8_t* image_array = scale_to_image(save_buffer, amount, width, height);
    //free(image_array);


    // Copy final result
    ColoredPoint* final_output = gpu_b;
    hipMemcpyAsync(host_points, final_output, amount * sizeof(Matrix<3, 1>), hipMemcpyDeviceToHost, stream);

    // Cleanup
    hipFreeAsync(gpu_a, stream);
    hipFreeAsync(gpu_b, stream);
    hipFreeAsync(d_states, stream);
    hipFreeAsync(gpu_floaty_save_buffer, stream);
    hipFreeAsync(gpu_save_buffer, stream);
    hipFreeAsync(scaling_data, stream);
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration_ms = end - start;
    *cpu_measured_time = duration_ms.count();
}


void create_triangle_gpu(ColoredPoint* points, int amount, int iterations) {
    ColoredPoint* gpu_points;
    ColoredPoint* gpu_buffer;
    hiprandState* gpu_rng_states;

    hipMalloc(&gpu_rng_states, amount * sizeof(hiprandState));

    hipMalloc(&gpu_points, amount * sizeof(ColoredPoint));
    hipMalloc(&gpu_buffer, amount * sizeof(ColoredPoint));

    hipMemcpy(gpu_points, points, amount * sizeof(ColoredPoint), hipMemcpyHostToDevice);
    
    
    unsigned long long kb = amount * sizeof(ColoredPoint) / 1024;
    unsigned long long mb = kb / 1024;
    printf("Size of copy: %llu KB\n", kb);
    printf("Size of copy: %llu MB\n", mb);
    int threadsPerBlock = 256;
    int blocksPerGrid = (amount + threadsPerBlock - 1) / threadsPerBlock;
    blocksPerGrid = min(blocksPerGrid, 65535); // Limit to 65535 blocks
    printf("blocksPerGrid: %d\n", blocksPerGrid);
    setup_rng_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_rng_states, amount, time(NULL));

    create_triangle_gpu_kernel<<<blocksPerGrid, threadsPerBlock>>>(gpu_points, amount, iterations, gpu_buffer, gpu_rng_states);
    hipDeviceSynchronize();
    hipMemcpy(points, gpu_buffer, amount * sizeof(ColoredPoint), hipMemcpyDeviceToHost);
    hipFree(gpu_points);
    hipFree(gpu_buffer);
    hipFree(gpu_rng_states);

}

Matrix<4, 4> create_random_affine_matrix_color() {
    Matrix<4, 4> scale_matrix;
    double data[4][4]= {
        {(double)stablerand_next(&stable_random) * 2 - 1, 0, 0, 0},
        {0, (double)stablerand_next(&stable_random) * 2 -1, 0, 0},
        {0, 0, (double)stablerand_next(&stable_random) * 2 - 1, 0},
        {0, 0, 0, 1}
    };

    /*double data[4][4]= {
        {1, 0, 0, 0},
        {0, 1, 0, 0},
        {0, 0, 1, 0},
        {0, 0, 0, 1}
    };*/
    scale_matrix.setData(data);
    Matrix<4, 4> translation_matrix;
    double translation_data[4][4] = {
        {1, 0, 0, (double)stablerand_next(&stable_random) *2 - 1},
        {0, 1, 0, (double)stablerand_next(&stable_random) *2 - 1},
        {0, 0, 1, (double)stablerand_next(&stable_random) *2 - 1},
        {0, 0, 0, 1}
    };
    translation_matrix.setData(translation_data);

    double angleYaw = (double)stablerand_next(&stable_random) * 2 * OUR_PI - OUR_PI;
    double anglePitch = (double)stablerand_next(&stable_random) * 2 * OUR_PI - OUR_PI;
    double angleRoll = (double)stablerand_next(&stable_random) * 2 * OUR_PI - OUR_PI;

    double yaw_data[4][4] = {
        {cos(angleYaw), -sin(angleYaw), 0, 0},
        {sin(angleYaw), cos(angleYaw), 0, 0},
        {0, 0, 1, 0},
        {0, 0, 0, 1}
    };

    double pitch_data[4][4] = {
        {cos(anglePitch), 0, sin(anglePitch), 0},
        {0, 1, 0, 0},
        {-sin(anglePitch), 0, cos(anglePitch), 0},
        {0, 0, 0, 1}
    };

    double roll_data[4][4] = {
        {1, 0, 0, 0},
        {0, cos(angleRoll), -sin(angleRoll), 0},
        {0, sin(angleRoll), cos(angleRoll), 0},
        {0, 0, 0, 1}
    };
    Matrix<4, 4> yaw_matrix;
    yaw_matrix.setData(yaw_data);
    Matrix<4, 4> pitch_matrix;
    pitch_matrix.setData(pitch_data);
    Matrix<4, 4> roll_matrix;
    roll_matrix.setData(roll_data);

    Matrix<4, 4> rotation_matrix = yaw_matrix * pitch_matrix * roll_matrix;


    return scale_matrix * rotation_matrix * translation_matrix;
}

template <size_t T>
Matrix<T, T>* random_matrix_cpu(Matrix<T, T>* firstMatrix, Matrix<T, T>* secondMatrix, Matrix<T,T>* thridMatrix) {
    int index = ((int)(stablerand_next(&stable_random) * 3)) % 3;

    if (index == 0) {
        return firstMatrix;
    }
    if (index == 1) {
        return secondMatrix;
    }
    return thridMatrix;
}



void create_triangle_cpu_single(ColoredPoint* host_points, int amount, int iterations,
    int width, int height, char* output_buffer, float* cpu_measured_time, Matrix<3, 3>* pos_random_matrixes, Matrix<4, 4>* color_random_matrixes) {
        auto start = std::chrono::high_resolution_clock::now();

        for (int i = 0; i < iterations; i++) {
            for (int j = 0; j < amount; j++) {
                ColoredPoint* point = host_points + j;
                Matrix<3, 3>* random_pos_matrix = random_matrix_cpu(&pos_random_matrixes[0], &pos_random_matrixes[1], &pos_random_matrixes[2]);
                Matrix<4, 4>* random_color_matrix = random_matrix_cpu(&color_random_matrixes[0], &color_random_matrixes[1], &color_random_matrixes[2]);
                point->pos = (*random_pos_matrix) * point->pos;
                point->color = (*random_color_matrix) * point->color;
            }
        }
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> duration_ms = end - start;
        *cpu_measured_time = duration_ms.count();
}


int main() {
    // 4321
    stablerand_init(&stable_random, 929290504577);
    //srand(4321);
    int width = 1000;
    int height = 1000;
    int image_size = width * height;


    Matrix<3, 3> randomMatrixOne = create_random_affine_matrix();
    Matrix<3, 3> randomMatrixTwo = create_random_affine_matrix();
    Matrix<3, 3> randomMatrixThree = create_random_affine_matrix();
    Matrix<4, 4> randomColorMatrixOne = create_random_affine_matrix_color();
    Matrix<4, 4> randomColorMatrixTwo = create_random_affine_matrix_color();
    Matrix<4, 4> randomColorMatrixThree = create_random_affine_matrix_color();
    //randomMatrixOne.print();
    //randomColorMatrixOne.print();
    //randomMatrixTwo.print();
    //randomColorMatrixThree.print();

    Matrix<3, 3> matrixesArray[3] = {randomMatrixOne, randomMatrixTwo, randomMatrixThree};
    hipMemcpyToSymbol(HIP_SYMBOL(global_matrixes_data), &matrixesArray, sizeof(Matrix<3, 3>) * 3);

    Matrix<4,4> colorMatrixesArray[3] = { randomColorMatrixOne, randomColorMatrixTwo, randomColorMatrixThree};
    hipMemcpyToSymbol(HIP_SYMBOL(global_matrixes_data), &colorMatrixesArray, sizeof(Matrix<4, 4>) * 3, sizeof(Matrix<3, 3>) * 3);


    /*char bigTestBuffer[sizeof(Matrix<3, 3>) * 3 + sizeof(Matrix<4, 4>) * 3];
    memcpy(bigTestBuffer, matrixesArray, sizeof(sizeof(Matrix<3, 3>) * 3));

    memcpy(bigTestBuffer + sizeof(Matrix<3, 3>) * 3, colorMatrixesArray, sizeof(Matrix<4, 4>) * 3);
    char* rawPtrToColorPart = bigTestBuffer + sizeof(Matrix<3, 3>) * 3;
    Matrix<4, 4>* colTest = (Matrix<4, 4>*)rawPtrToColorPart;
    printf("printing color matrices\n");
    colTest[0].print();
    colTest[1].print();
    colTest[2].print();
    printf("printing color matrices done\n");*/



    // Generate random points
    int amount = 400000;
    //printf("Generating random points...\n");
    ColoredPoint* points = generate_random_points(amount);
    //points[0].color.print();

    ColoredPoint* buffer = (ColoredPoint*)malloc(amount * sizeof(ColoredPoint));
    char* outputBuffer = (char*)malloc(width * height * 3 * sizeof(char));

    //printf("done\n");
    printf("Creating triangle...\n");
    //create_triangle(points, amount, 200, buffer, &bottomLeftMatrix, &bottomRightMatrix, &topMatrix);
    // create_triangle_gpu(points, amount, 20, bottomLeftMatrix, bottomRightMatrix, topMatrix);
    create_triangle_gpu_with_frames(points, amount, 200, width, height);
    float unusedTime = 0;
    create_triangle_gpu_single(points, amount, 200, width, height, outputBuffer, &unusedTime);
    printf("Amount of iterations, Amount of points, Time GPU, Time CPU\n");

    /*for (int k = 10; k < 400; k += 20) {
        ColoredPoint* localPoints = generate_random_points(k);
        ColoredPoint* localBuffer = (ColoredPoint*)malloc(k * sizeof(ColoredPoint));
        ColoredPoint* clonedPointsForMeasureGPU = (ColoredPoint*) malloc(k * sizeof(ColoredPoint));
        ColoredPoint* clonedPointsForMeasureCPU = (ColoredPoint*) malloc(k * sizeof(ColoredPoint));

        for (int i = 10; i < 1000; i += 50) {

            float gpuTimeSum = 0;
            float cpuTimeSum = 0;
            for (int j = 0; j < 10; j++) {
                memcpy(clonedPointsForMeasureCPU, localPoints, k * sizeof(ColoredPoint));
                memcpy(clonedPointsForMeasureGPU, localPoints, k * sizeof(ColoredPoint));
                float gpuTime = 0;
                float cpuTime = 0;
                create_triangle_gpu_single(clonedPointsForMeasureCPU, k, i, width, height, outputBuffer, &gpuTime);
                create_triangle_cpu_single(clonedPointsForMeasureGPU, k, i, width, height, outputBuffer, &cpuTime, &matrixesArray[0], &colorMatrixesArray[0]);
                gpuTimeSum += gpuTime;
                cpuTimeSum += cpuTime;
            }
            gpuTimeSum /= 10;
            cpuTimeSum /= 10;
    
            printf("%d, %d, %f, %f\n", i, k, gpuTimeSum, cpuTimeSum);
    
        }
        free(localBuffer);
        free(clonedPointsForMeasureGPU);
        free(clonedPointsForMeasureCPU);
        free(localPoints);
    }*/

    save_image_with_name((uint8_t*) outputBuffer, width, height, "output.ppm");
    
    // Save the image
    //save_black_white_image(image_array, width, height);

    system("ffmpeg -y -framerate 5 -i ./vid_imgs/frame_%03d.ppm -c:v libx264 -pix_fmt yuv420p output.mp4");
    
    // Free the memory
    free(outputBuffer);
    free(points);
    free(buffer);
    
    return 0;
}